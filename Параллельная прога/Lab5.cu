#include "hip/hip_runtime.h"
%%writefile Lab5.cu
#include <iostream>
#include <hip/hip_runtime.h>

#define N 16

__global__ void transposeKernel(float *in, float *out, int n) {
    __shared__ float tile[16][16];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < n && y < n) {
        tile[threadIdx.y][threadIdx.x] = in[y * n + x];
    }
    __syncthreads();

    x = blockIdx.y * blockDim.x + threadIdx.x;
    y = blockIdx.x * blockDim.y + threadIdx.y;

    if (x < n && y < n) {
        out[y * n + x] = tile[threadIdx.x][threadIdx.y];
    }
}

void initMatrix(float *matrix, int n) {
    for (int i = 0; i < n * n; i++) {
        matrix[i] = static_cast<float>(i);
    }
}

void printMatrix(const float *matrix, int n) {
    for (int i = 0; i < 16; ++i) {
        for (int j = 0; j < 16; ++j) {
            std::cout << matrix[i * n + j] << " ";
        }
        std::cout << std::endl;
    }
}

int main() {
    float *h_in, *h_out;
    float *d_in, *d_out;

    size_t size = N * N * sizeof(float);
    
    h_in = (float*)malloc(size);
    h_out = (float*)malloc(size);

    initMatrix(h_in, N);
    printMatrix(h_in, N);
    std::cout << std::endl;

    hipMalloc((void**)&d_in, size);
    hipMalloc((void**)&d_out, size);

    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 grid(N / 16, N / 16);


    // Создание событий для замера времени
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    hipEventRecord(startEvent, 0);

    transposeKernel<<<grid, threads>>>(d_in, d_out, N);

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);


    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

    printMatrix(h_out, N);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
    cout << "\nTime: " << elapsedTime << " ms" << endl;

    hipFree(d_in);
    hipFree(d_out);
    free(h_in);
    free(h_out);

    return 0;
}







